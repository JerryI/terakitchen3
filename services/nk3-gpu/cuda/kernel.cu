#include "hip/hip_runtime.h"
__device__ void movingAverage (
    float4 * src, float * dest
) {
    if (threadIdx.x > 1024 - 2) return;

    float* _dest = &dest[ threadIdx.x << 3];
    float* _next = &dest[(threadIdx.x + 1) << 3];

    _dest[1] = (_next[1] + _dest[1])/2.0f;
    _dest[2] = (_next[2] + _dest[2])/2.0f;
}

__device__  void subtractFP (
    float4 * src, float * dest
) {
    float4 params;
        params.x = dest[(1024 << 3)];     //n0
        params.y = dest[(1024 << 3) + 1]; //thickness
        params.z = dest[(1024 << 3) + 2]; //scale
        params.w = dest[(1024 << 3) + 3];
    //[0] - n0
    //[1] - thickness
    //[2] - scale
    //[3] - phase shift
    //
    float4 one = src[threadIdx.x];

    //scaling
    one.y = one.y * params.z;

    float* _dest = &dest[threadIdx.x << 3];

    const float thickness = params.y;

    //optimised using Experimental`OptimizedExpression on Wolfram Language
    const float n    = _dest[1];
    const float k    = _dest[2];

    const float freq = _dest[0];

    const float CONSTF = 6.28331f * thickness;

    float var40 = k*k;
    float var41 = 1.0f + n;
    float var54 = var41*var41;
    float var57 = var40 + var54;
    float var85 = n*n; 
    float var90 = 2.0f*CONSTF*freq*n;
    float var81 = -2.0f*CONSTF*freq*k;
    float var82 = expf(var81);
    float var58 = 1.0f/(var57*var57);
    float var93 = -1.0f + var40 + var85; 
    float var91 = cosf(var90);
    float var83 = -2.0f + k;
    float var84 = var83*k;
    float var86 = -1.0f + var84 + var85;
    float var87 = 2.0f + k;
    float var88 = k*var87;
    float var89 = -1.0f + var88 + var85;
    float var94 = sinf(var90);
    float var80 = var57*var57;
    float var99 = (var40 + (-1.0f + n)*(-1.0f + n));
    float var98 = expf(2.0f*CONSTF*freq*k);

    float var92 = 4.0f*k*var93*var94;

    float abs   = sqrtf(var58*(var99*var99/(var98*var98) + var80 + var82*(-2.0f*var86*var89*var91 + 2.0f*var92)));

    float arg = atan2f(var82*var58*(4.0f*k*var93*var91 + var86*var89*var94),var82*var58*(var98*var80 - var86*var89*var91 + var92));    

    if (!isfinite(abs) || !isfinite(arg)) {
        abs = 1.0f; arg = 0.0f;
    }

    if (abs < 0.0f) {abs = 1.0f; arg = 0.0f;};

    //debug
    _dest[5] = abs;
    _dest[6] = arg;

    if (one.y * abs > 1.1f) {
        _dest[3] = 1.0f;
        _dest[4] = 0.0f;
    } else {
        _dest[3] = one.y * abs;
        _dest[4] = one.z - arg + params.w;
    }
}

__device__ void solveNK (
    float4 * src, float * dest
) {
    float4 params;
        params.x = dest[(1024 << 3)];
        params.y = dest[(1024 << 3) + 1];
        params.z = dest[(1024 << 3) + 2];
        params.w = dest[(1024 << 3) + 3];

    float* _dest = &dest[threadIdx.x << 3];

    const float n0 = params.x;
    const float thickness = params.y;

    //_dest[0] - freqs
    //_dest[1] - n
    //_dest[2] - k
    //_dest[3] - t abs
    //_dest[4] - t ph       
    
    const float fT = 1.0f/(thickness * _dest[0]);
    const float logT = logf(_dest[3]);
    const float ph   = _dest[4];

    float np = _dest[1];
    float kp = _dest[2];

    float n = 0;
    float k = 0;
    float denominator;
    float arg;
    float modulus;
    float im, re, n2;
    
    for(int i=0; i<30; ++i) {
        n2 = (1.0f + np);
        n2 = n2 * n2;
        denominator = 1.0f/(kp*kp + n2);
        denominator = denominator * denominator;

        re = denominator*(np*n2 + kp*kp*(2.0f + np));
        im = denominator*(kp*(kp*kp + np*np - 1.0f));

        modulus = sqrtf(re*re + im*im);
        arg = atan2f(im, re);

        n = 1.0f + 0.159152f * fT * (ph - arg);
        k = - 0.159152f * fT * (logT - logf(4.0f*modulus));

        np = n; kp = k;
    }

    if (!isfinite(n) || !isfinite(k) || n < 0.0f) {
        n = n0;
        k = 0.0f;
    }

    _dest[1] = n;
    _dest[2] = k;
}

__global__ void k_solveNK (
    float4 * src, float * dest
) {
    solveNK(src, dest);
}

__global__ void k_subtractFP (
    float4 * src, float * dest
) {
    subtractFP(src, dest);
}

__global__ void k_movingAverage (
    float4 * src, float * dest
) {
    movingAverage(src, dest);
}

__global__ void initialise (
    float4 * src, float * dest
) {
    //__shared__ float4 transmission[1000];
    if (threadIdx.x > 1023) return;

    const float4  params = src[1024];
    float4 one    = src[threadIdx.x];
    //one.x - freq, 
    //one.y - abs, 
    //one.z - ph
    const float n0 = params.x;
    const float thickness = params.y;
    
    const float fT = 1.0f/(thickness * one.x);
    const float logT = logf(one.y);

    float* _dest = &dest[threadIdx.x << 3];

    if (threadIdx.x + blockIdx.x * blockDim.x == 0) {
        _dest[1024 << 3]     = params.x;
        _dest[(1024 << 3) + 1] = params.y;
        _dest[(1024 << 3) + 2] = params.z;
        _dest[(1024 << 3) + 3] = params.w;
    }

    float n = 1.0f + (0.159152f * (one.z + params.w) * fT);
    float k = - (0.159152f * logT * fT);

    if (!isfinite(n))
        n = n0;
    
    if (!isfinite(k))
        k = 0.0f;    

    _dest[0] = one.x;
    _dest[1] = n;
    _dest[2] = k;

    _dest[3] = one.y;
    _dest[4] = one.z + params.w;

}

__global__ void generateTDS (
    float4 * src, 
    float * dest, 
    mint cycles_dry, 
    mint cycles_wet,
    float2 *dataset
) {

}    

__device__ void cpyDest (
    float * dest_local, float * dest
) {
    const int shift = blockIdx.x * (1025 * 8);

    dest[(threadIdx.x << 3) + shift] = dest_local[(threadIdx.x << 3)];
    dest[(threadIdx.x << 3) + shift + 1] = dest_local[(threadIdx.x << 3) + 1];
    dest[(threadIdx.x << 3) + shift + 2] = dest_local[(threadIdx.x << 3) + 2];
    dest[(threadIdx.x << 3) + shift + 3] = dest_local[(threadIdx.x << 3) + 3];
    dest[(threadIdx.x << 3) + shift + 4] = dest_local[(threadIdx.x << 3) + 4];
    dest[(threadIdx.x << 3) + shift + 5] = dest_local[(threadIdx.x << 3) + 5];
    dest[(threadIdx.x << 3) + shift + 6] = dest_local[(threadIdx.x << 3) + 6];
    dest[(threadIdx.x << 3) + shift + 7] = dest_local[(threadIdx.x << 3) + 7];    
}

__global__ void autorun (
    float4 * src, 
    float * dest, 
    mint cycles_dry, 
    mint cycles_wet,
    float3 *dataset
) {
    __shared__ float _local_dest[1024*8 + 8];

    const float4  params = src[1024];
    float4 one    = src[threadIdx.x];
    //one.x - freq, 
    //one.y - abs, 
    //one.z - ph
    const float n0 = params.x;
    const float thickness = dataset[blockIdx.x].x;
    const float scale = dataset[blockIdx.x].y;
    const float phase = dataset[blockIdx.x].z;
    
    const float fT = 1.0f/(thickness * one.x);
    const float logT = logf(one.y * scale);

    float* _dest = &_local_dest[threadIdx.x << 3];

    if (threadIdx.x == 0) {
        _dest[1024 << 3]       = params.x;
        _dest[(1024 << 3) + 1] = dataset[blockIdx.x].x;
        _dest[(1024 << 3) + 2] = dataset[blockIdx.x].y;
        _dest[(1024 << 3) + 3] = phase;
    }

    float n = 1.0f + (0.159152f * (one.z + phase) * fT);
    float k = - (0.159152f * logT * fT);

    if (!isfinite(n))
        n = n0;
    
    if (!isfinite(k))
        k = 0.0f;    

    _dest[0] = one.x;
    _dest[1] = n;
    _dest[2] = k;

    _dest[3] = one.y * scale;
    _dest[4] = one.z + phase;

    __syncthreads();

    solveNK(src, _local_dest);

    for (int i=0; i<cycles_dry; ++i) { 
      subtractFP(src, _local_dest);
      solveNK(src, _local_dest);
    }

    for (int i=0; i<cycles_wet; ++i) { 
      subtractFP(src, _local_dest);
      solveNK(src, _local_dest);
      __syncthreads();
      movingAverage(src, _local_dest);
    } 

    __syncthreads();
    cpyDest(_local_dest, dest);

}