#include "hip/hip_runtime.h"

__global__  void subtractFP (
    float4 * src, float * dest, mint len
) {
    const float4 params = src[1000];
    float* _dest = &dest[threadIdx.x << 3];

    const float thickness = params.y;

    float4 one    = src[threadIdx.x];

    //optimised using Experimental`OptimizedExpression on Wolfram Language
    const float n    = _dest[1];
    const float k    = _dest[2];

    const float freq = _dest[0];

    const float CONSTF = 6.28331f * thickness;

    float var40 = k*k;
    float var41 = 1.0f + n;
    float var54 = var41*var41;
    float var57 = var40 + var54;
    float var85 = n*n; 
    float var90 = 2.0f*CONSTF*freq*n;
    float var81 = -2.0f*CONSTF*freq*k;
    float var82 = expf(var81);
    float var58 = 1.0f/(var57*var57);
    float var93 = -1.0f + var40 + var85; 
    float var91 = cosf(var90);
    float var83 = -2.0f + k;
    float var84 = var83*k;
    float var86 = -1.0f + var84 + var85;
    float var87 = 2.0f + k;
    float var88 = k*var87;
    float var89 = -1.0f + var88 + var85;
    float var94 = sinf(var90);
    float var80 = var57*var57;
    float var99 = (var40 + (-1.0f + n)*(-1.0f + n));
    float var98 = expf(2.0f*CONSTF*freq*k);

    float var92 = 4.0f*k*var93*var94;

    float abs   = sqrtf(var58*(var99*var99/(var98*var98) + var80 + var82*(-2.0f*var86*var89*var91 + 2.0f*var92)));

    float arg = atan2f(var82*var58*(4.0f*k*var93*var91 + var86*var89*var94),var82*var58*(var98*var80 - var86*var89*var91 + var92));    

    if (!isfinite(abs) || !isfinite(arg)) {
        abs = 1.0f; arg = 0.0f;
    }

    if (abs < 0.0f) {abs = 1.0f; arg = 0.0f;};

    //debug
    _dest[5] = abs;
    _dest[6] = arg;

    _dest[3] = one.y * abs;
    _dest[4] = one.z - arg;
}

__global__ void solveNK (
    float4 * src, float * dest, mint len
) {
    const float4 params = src[1000];
    float* _dest = &dest[threadIdx.x << 3];

    const float n0 = params.x;
    const float thickness = params.y;

    //_dest[0] - freqs
    //_dest[1] - n
    //_dest[2] - k
    //_dest[3] - t abs
    //_dest[4] - t ph       
    
    const float fT = 1.0f/(thickness * _dest[0]);
    const float logT = logf(_dest[3]);
    const float ph   = _dest[4];

    float np = _dest[1];
    float kp = _dest[2];

    float n = 0;
    float k = 0;
    float denominator;
    float arg;
    float modulus;
    float im, re, n2;
    
    for(int i=0; i<30; ++i) {
        n2 = (1.0f + np);
        n2 = n2 * n2;
        denominator = 1.0f/(kp*kp + n2);
        denominator = denominator * denominator;

        re = denominator*(np*n2 + kp*kp*(2.0f + np));
        im = denominator*(kp*(kp*kp + np*np - 1.0f));

        modulus = sqrtf(re*re + im*im);
        arg = atan2f(im, re);

        n = 1.0f + 0.159152f * fT * (ph - arg);
        k = - 0.159152f * fT * (logT - logf(4.0f*modulus));

        np = n; kp = k;
    }

    if (!isfinite(n))
        n = n0;
    
    if (!isfinite(k))
        k = 0.0f;
    
    _dest[1] = n;
    _dest[2] = k;
}

__global__ void movingAverage (
    float4 * src, float * dest, mint len
) {
    float* _dest1 = &dest[(threadIdx.x << 2)];
    float* _dest2 = &dest[(threadIdx.x << 2) + (1 >> 3)];

    _dest2[1] = (_dest2[1] + _dest1[1])*0.5f;
    _dest2[2] = (_dest2[2] + _dest1[2])*0.5f;

    __syncthreads();

    _dest1[1] = (_dest2[1] + _dest1[1])*0.5f;
    _dest1[2] = (_dest2[2] + _dest1[2])*0.5f;
}

__global__ void initialise (
    float4 * src, float * dest, mint len
) {
    //__shared__ float4 transmission[1000];
    if (threadIdx.x > len) return;

    const float4 params = src[1000];
    float4 one    = src[threadIdx.x];
    //one.x - freq, 
    //one.y - abs, 
    //one.z - ph
    const float n0 = params.x;
    const float thickness = params.y;
    
    const float fT = 1.0f/(thickness * one.x);
    const float logT = logf(one.y);

    float* _dest = &dest[threadIdx.x << 3];

    float n = 1.0f + (0.159152f * (one.z + params.w) * fT);
    float k = - (0.159152f * logT * fT);

    if (!isfinite(n))
        n = n0;
    
    if (!isfinite(k))
        k = 0.0f;    

    _dest[0] = one.x;
    _dest[1] = n;
    _dest[2] = k;

    _dest[3] = one.y;
    _dest[4] = one.z + params.w;

}